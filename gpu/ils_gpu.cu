#include "hip/hip_runtime.h"
#include "sspemdd_sequential.h"
#include "sspemdd_utils.h"
#include "assert.h"
#include "hip/hip_runtime_api.h"
#include <cmath>

#define BLOCKSIZE 8

#define MAX_MAT_SIZE 2048
#define MAX_FREQS 1000
#define MAX_INTERFACES 10
#define MAX_WNUMS 100
#define ORD_RICH 1

#include "bisect_gpu.h"

//#define m_CopyToGPU(d, s,  bytes){ if(hipSuccess != hipMalloc((void**) &d, bytes)) boinc_temporary_exit(60); checkCudaErrors(hipMemcpy((void*) d, (void*)s, bytes, hipMemcpyHostToDevice)); }

#define m_CopyToGPU2(s, elements, type)\
	type* g_##s;\
	checkCudaErrors(hipMalloc((void**) &g_##s, elements*sizeof(type)));\
	checkCudaErrors(hipMemcpy((void*) g_##s, (void*)s, elements*sizeof(type), hipMemcpyHostToDevice));

#define m_FreeHostAndGPU(s)\
	free(s);\
	checkCudaErrors(hipFree(g_##s));\

__device__ void FillLocalArrays (
		const int tid,
		const float cb,
		const float rhob, 
		const int batch_sz, 
		const int cws_sz, 
		const float* cws, 
		float rhos[],
		float c1s[],
		float c2s[])
{
	// FIXME: magic constant!
	for (int i = 0; i < cws_sz +1; ++i)
		c1s[i] = 1500;

	for (unsigned i = 0; i < cws_sz - 1; i++)
	{
		c1s[i] = cws[i*batch_sz + tid];
		c2s[i] = cws[(i + 1)*batch_sz + tid];
		rhos[i] = 1;
	}

	c1s[cws_sz - 1] = cws[(cws_sz - 1)*batch_sz + tid];
	c2s[cws_sz - 1] = cws[(cws_sz - 1)*batch_sz + tid];
	rhos[cws_sz - 1] = 1;

	c1s[cws_sz] = cb;
	c2s[cws_sz] = cb;
	rhos[cws_sz] = rhob;
}

__device__ void FillDiagonals(
		const float omega,
		const float c[], 
		const int c_sz,
		const float rho[],
		const int interface_idcs[], 
		const int interface_idcs_sz,
		const float meshsizes[],
		float md[], 
		float ud[] /*sd*/, 
		int& mat_size)
{
	int N_points = c_sz;
	int layer_number = 0;

	float ld[MAX_MAT_SIZE];
	float dz = meshsizes[layer_number];
	for (int i = 0; i < N_points - 2; i++)
	{
		if ((layer_number < interface_idcs_sz) && (i == (interface_idcs[layer_number]-1)))
		{
			// special case of the point at the interface
			++layer_number;
			float dz_next = meshsizes[layer_number];
			float cp = c[i + 1];
			float dp = rho[i + 1];
			float cm = c[i];
			float dm = rho[i];
			float q = 1 / (dz_next * dm + dz * dp);

			ld[i] = 2 * q * dp / dz;
			// Magic!
			md[i] = -2 * q * (dz_next * dp + dz * dm) / (dz * dz_next) +
						omega * omega * q * (dz * dp * cp * cp + dz_next * dm * cm * cm) /
							(cp * cp * cm * cm);
			ud[i] = 2 * q * dm / dz_next;
			dz = dz_next;
		} 
		else 
		{
			// ordinary point
			ud[i] = (1 / (dz * dz)); 
			ld[i] = ud[i];
			md[i] = (-2 / (dz * dz) + omega * omega / (c[i + 1] * c[i + 1]));
		}
	}

	// TODO: merge me with ud-ld-md cycle and remove ld array!
	// Symmetrize the matrix
	for (int i = 0; i < N_points - 3; i++)
		ud[i] = sqrt(ud[i] * ld[i + 1]);
	mat_size = N_points - 2;
	// DIAGONALS!!!
}

__device__ Interval ComputeWavenumsLimits(
		const float omega, 
		const float c[], 
		const int c_sz)
{
	float cmin = c[0], cmax = c[0];
	for (int i = 0; i < c_sz; i++)
	{
		if (c[i] < cmin)
			cmin = c[i];
		if (c[i] > cmax)
			cmax = c[i];
	}
	float kappamax = omega / cmin;
	float kappamin = omega / cmax;
	return Interval {kappamin*kappamin, kappamax*kappamax};
}

__device__ void FillLayers(const int rr, 
		const int n_layers,
		const float* depths, 
		const float* rhos, 
		const float* c1s,
		const float* c2s, 
		const int* Ns_points, 
		float mesh[], 
		int interface_idcs[], int& interface_idcs_sz,
		float c[], int& c_sz, float rho[])
{
	c[0] = 0;
	rho[0] = 0;

	// TODO: Rewrite me, i am UGLY ((
	int n = 1; //total number of points
	float zp = 0;
	for (unsigned i = 0; i < n_layers; ++i)
	{
		int n_points_layer = Ns_points[i] * rr;
		float zc = depths[i];
		mesh[i] = (zc - zp) / n_points_layer; // dz

		c[n - 1] = c1s[i];
		rho[n - 1] = rhos[i];

		for (unsigned k = 1; k <= n_points_layer; ++k)
		{
			rho[n] = rhos[i];
			c[n] = (c1s[i] + (c2s[i] - c1s[i]) * k / n_points_layer);
			++n;
		}
		if (i < n_layers - 1)
			interface_idcs[i] = n - 1;
		zp = zc;
	}

	interface_idcs_sz = n_layers - 1;
	c_sz = n;
}

__device__ void ComputeWavenums(
		const float omega,
		const int n_layers,
		const int* Ns_points,
		const float* depths,
		const float rhos[],
		const float c1s[],
		const float c2s[],
		float wnums[],
		int& wnums_sz)
{
	// Strange things happen here...
	int  Ns_points_aligned [MAX_MAT_SIZE];
	for (int i = 0; i < n_layers; ++i)
		Ns_points_aligned[i] = 12 * (Ns_points[i] / 12);

	float coeff_extrap[4][4] = {
			{1,0,0,0},
			{-1, 2, 0, 0},
			{0.5, -4, 4.5, 0},
			{-1 / float(6), 4, -13.5, 32 / float(3)}};

	for (int rr = 1; rr <= ORD_RICH; ++rr)
	{
		float mesh [MAX_MAT_SIZE];
		int interface_idcs [MAX_INTERFACES]; 
		int interface_idcs_sz;
		float c [MAX_MAT_SIZE];
		int   c_sz;
		float rho [MAX_MAT_SIZE];
		FillLayers(rr, n_layers, depths, rhos, c1s, c2s, Ns_points_aligned, 
				mesh, interface_idcs, interface_idcs_sz, c, c_sz, rho);

		int mat_size;
		float md [MAX_MAT_SIZE];
		float sd [MAX_MAT_SIZE];
		FillDiagonals(omega, c, c_sz, rho, interface_idcs, interface_idcs_sz, mesh, 
				md, sd, mat_size);

		float wnums_rr [MAX_WNUMS];
		int wnums_rr_sz;
		Interval lim = ComputeWavenumsLimits(omega, c, c_sz);
		wnums_rr_sz = bisectGPU(md, sd, mat_size, lim.ll, lim.rl, wnums_rr);
		if (rr == 1) 
			wnums_sz = wnums_rr_sz;
		for (int i = 0; i < wnums_rr_sz; ++i)
			wnums[i] += (wnums_rr[i] * coeff_extrap[ORD_RICH-1][rr-1]);
	}
}

// This procedure computes MGV for a _single_ frequency
__device__ void ComputeModalGroupVelocities (
		const float freq,
		const int n_layers,
		const int* Ns_points,
		const float* depths,
		const float rhos[],
		const float c1s[],
		const float c2s[],
		float mgv[MAX_WNUMS],
		int& mgv_sz)
{
	float wnums1 [MAX_WNUMS] = {0}; int wnums1_sz;
	float wnums2 [MAX_WNUMS] = {0}; int wnums2_sz;
	// magic number for numerical differentiation procedure
	float deltaf = 0.05;
	float omega1 = 2 * LOCAL_M_PI * freq + deltaf;
	float omega2 = 2 * LOCAL_M_PI * freq;
	
	ComputeWavenums(omega1, n_layers, Ns_points, depths, rhos, c1s, c2s, wnums1, wnums1_sz);
	ComputeWavenums(omega2, n_layers, Ns_points, depths, rhos, c1s, c2s, wnums2, wnums2_sz);

	// Since with increase of omega the number of wave numbers
	// can only increase,  wnum2_sz <= wnum1_sz
	for (int i = 0; i < wnums2_sz; ++i)
		mgv[i] = (omega1 - omega2) / (sqrt(wnums1[i]) - sqrt(wnums2[i]));
	mgv_sz = wnums2_sz;
}

__global__ void EvalPoint_gpukernel(
		const int cws_sz, 
		const int dmaxsz,
		const float* cws, 
		const int* Ns_points,
		const float* depths,
		const float R, 
		const float tau, 
		const float rhob, 
		const float cb, 
		const float* freqs, 
		const int freqs_sz,
		const float* exp_delays,
		const int* exp_delays_sz,
		float* residual,
		int* n_res_global)
{
	int n_layers = cws_sz+1;
	
	const unsigned int tid = (blockIdx.x << BLOCKSIZE) + threadIdx.x;

	if (tid >= freqs_sz)
		return;
	float rhos[MAX_MAT_SIZE];
	float c1s[MAX_MAT_SIZE];
	float c2s[MAX_MAT_SIZE];
	FillLocalArrays(0, cb, rhob, 1, cws_sz, cws,  
			rhos, c1s, c2s);

	int n_residuals = 0;
	float residuals_local = 0;
	// Compute mgvs for all frequencies
	//assert (freqs_sz < MAX_FREQS);
	float calc_mgv[MAX_WNUMS];
	int calc_mgv_sz;
	ComputeModalGroupVelocities(freqs[tid], n_layers, Ns_points, depths, rhos, c1s, c2s, 
		calc_mgv, calc_mgv_sz);

	int min_size = calc_mgv_sz < exp_delays_sz[tid] ? 
		calc_mgv_sz : exp_delays_sz[tid];

	for (int j = 0; j < min_size; ++j) //iterate over modal velocities
	{
		float exp_delay = exp_delays[tid*dmaxsz + j];
		float calc_delay = R / calc_mgv[j];
		if (exp_delay > 0)
		{
			residuals_local += pow(exp_delay + tau - calc_delay, 2);
			++n_residuals;
		}
	}

	atomicAdd(residual, residuals_local);
	atomicAdd(n_res_global, n_residuals);

	//residual = sqrt(residuals_local/n_residuals);
}
__global__ void EvalPoints_gpukernel(
		const int batch_sz, 
		const int cws_sz, 
		const int dmaxsz,
		const float* cws, 
		const int* Ns_points,
		const float* depths,
		const float* R, 
		const float* tau, 
		const float* rhob, 
		const float* cb, 
		const float* freqs, 
		const int freqs_sz,
		//const float exp_delays[freqs_sz][dmaxsz], 
		const float* exp_delays,
		const int* exp_delays_sz,
		float* residuals)
{
	int n_layers = cws_sz+1;
	
	const unsigned int tid = (blockIdx.x << BLOCKSIZE) + threadIdx.x;

	float rhos[MAX_MAT_SIZE];
	float c1s[MAX_MAT_SIZE];
	float c2s[MAX_MAT_SIZE];
	FillLocalArrays(tid, cb[tid], rhob[tid], batch_sz, cws_sz, cws,  
			rhos, c1s, c2s);

	int n_residuals = 0;
	float residuals_local = 0;
	// Compute mgvs for all frequencies
	//assert (freqs_sz < MAX_FREQS);
	for (int i = 0; i < freqs_sz; ++i)
	{
		float calc_mgv[MAX_WNUMS];
		int calc_mgv_sz;
		ComputeModalGroupVelocities(freqs[i], n_layers, Ns_points, depths, rhos, c1s, c2s, 
			calc_mgv, calc_mgv_sz);

		int min_size = calc_mgv_sz < exp_delays_sz[i] ? 
			calc_mgv_sz : exp_delays_sz[i];

		for (int j = 0; j < min_size; ++j) //iterate over modal velocities
		{
			float exp_delay = exp_delays[i*dmaxsz + j];
			float calc_delay = R[tid] / calc_mgv[j];
			if (exp_delay > 0)
			{
				residuals_local += pow(exp_delay + tau[tid] - calc_delay, 2);
				++n_residuals;
			}
		}

	}
	residuals[tid] = sqrt(residuals_local/n_residuals);
}

void EvalPointGPU(
		search_space_point &point,
		const std::vector<double> &freqs_d,
		const std::vector<unsigned> &Ns_points_d,
		const std::vector<double> &depths_d,
		const std::vector<std::vector<double>> &modal_delays)
{
	// Transform AoS to SoA
	size_t cws_sz = point.cws.size();
	float *cws = (float*) malloc(cws_sz*sizeof(float));
	for (size_t i = 0; i < cws_sz; ++i)
		cws[i] = point.cws[i];
	//TODO: stop converting this data every time
	assert (freqs_d.size() == modal_delays.size());
	
	//std::cout << " copy const" << std::endl;
	// freqs array
	int freqs_sz = freqs_d.size();
	//std::cout << " num freqs " << freqs_sz << std::endl;
	float *freqs = (float*) malloc(freqs_sz*sizeof(float));
	for (int i = 0; i < freqs_sz; ++i)
		freqs[i] = freqs_d[i];

	// exp_delays_sz
	int *exp_delays_sz = (int*) malloc(freqs_sz*sizeof(int));
	for (size_t i = 0; i < freqs_sz; ++i)
		exp_delays_sz[i] = modal_delays[i].size();

	// exp_delays 2d array
	int dmaxsz = 0;
	for (size_t i = 0; i < freqs_sz; ++i)
		dmaxsz = std::max(dmaxsz, exp_delays_sz[i]);
	float *exp_delays = (float*) malloc(dmaxsz*freqs_sz*sizeof(float));
	for (size_t i = 0; i < modal_delays.size(); ++i)
		for (size_t j = 0; j < modal_delays[i].size(); ++j)
			exp_delays[i*dmaxsz + j] = modal_delays[i][j];

	int n_layers = depths_d.size();
	float *depths = (float*) malloc(n_layers*sizeof(float));
	for (int i=0; i<n_layers; ++i)
		depths[i] = depths_d[i];

	int *Ns_points = (int*) malloc(n_layers*sizeof(int));
	for (int i=0; i<n_layers; ++i)
		Ns_points[i] = Ns_points_d[i];

	// output array
	float *residual = (float*) malloc(sizeof(float));
	residual[0] = 0;
	int *n_res_global = (int*) malloc(sizeof(int));
	n_res_global[0] = 0;

	m_CopyToGPU2(cws, cws_sz, float);
	m_CopyToGPU2(freqs, freqs_sz, float);
	m_CopyToGPU2(exp_delays, dmaxsz*freqs_sz, float);
	m_CopyToGPU2(exp_delays_sz, freqs_sz, int);
	m_CopyToGPU2(depths, n_layers, float);
	m_CopyToGPU2(Ns_points, n_layers, int);
	m_CopyToGPU2(residual, 1, float);
	m_CopyToGPU2(n_res_global, 1, int);

	hipEvent_t kernel_start, kernel_stop;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	hipEventRecord(kernel_start);

	EvalPoint_gpukernel <<< freqs_sz/(1<<BLOCKSIZE), 1<<BLOCKSIZE >>> 
		(cws_sz, 
		 dmaxsz, 
		 g_cws, 
		 g_Ns_points, 
		 g_depths, 
		 point.R, 
		 point.tau, 
		 point.rhob, 
		 point.cb, 
		 g_freqs, 
		 freqs_sz,
		 g_exp_delays,
		 g_exp_delays_sz,
		 g_residual,
		 g_n_res_global);

	#ifndef NDEBUG
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkCudaErrors(err);
	//printf("\n Bla");
	#endif

	hipDeviceSynchronize();
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	float runTime;
	hipEventElapsedTime(&runTime, kernel_start, kernel_stop);


	float tm = runTime / 1000;
	//printf("\n Time: %f", tm);
	
	checkCudaErrors(hipMemcpy((void*) residual, (void*)g_residual, 
				sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void*) n_res_global, (void*)g_n_res_global, 
				sizeof(int), hipMemcpyDeviceToHost));
	//printf("\n Res_loc: %f %i", *residual, *n_res_global);
	point.residual = std::sqrt(*residual / *n_res_global);

	m_FreeHostAndGPU(Ns_points);
	m_FreeHostAndGPU(depths);
	m_FreeHostAndGPU(exp_delays_sz);
	m_FreeHostAndGPU(exp_delays);
	m_FreeHostAndGPU(freqs);
	m_FreeHostAndGPU(cws);
	m_FreeHostAndGPU(residual);
	m_FreeHostAndGPU(n_res_global);
}
void EvalPointBatchGPU(
		std::vector <search_space_point> &batch,
		const std::vector<double> &freqs_d,
		const std::vector<unsigned> &Ns_points_d,
		const std::vector<double> &depths_d,
		const std::vector<std::vector<double>> &modal_delays)
{
	// Transform AoS to SoA
	size_t sz = batch.size();
	size_t cws_sz = batch[0].cws.size();
	float *R =   (float*) malloc(sz*sizeof(float));
	float *tau = (float*) malloc(sz*sizeof(float));
	float *rhob= (float*) malloc(sz*sizeof(float));
	float *cb =  (float*) malloc(sz*sizeof(float));
	float *cws = (float*) malloc(sz*cws_sz*sizeof(float));
	for (size_t i = 0; i < sz; ++i)
	{
		R[i]    = batch[i].R;
		tau[i]  = batch[i].tau;
		rhob[i] = batch[i].rhob;
		cb[i]   = batch[i].cb;

		for (size_t j = 0; j < cws_sz; ++j)
			cws[j*sz + i] = batch[i].cws[j];
	}
	//TODO: stop converting this data every time
	assert (freqs_d.size() == modal_delays.size());
	
	std::cout << " copy const" << std::endl;
	// freqs array
	int freqs_sz = freqs_d.size();
	std::cout << " num freqs " << freqs_sz << std::endl;
	float *freqs = (float*) malloc(freqs_sz*sizeof(float));
	for (int i = 0; i < freqs_sz; ++i)
		freqs[i] = freqs_d[i];

	// exp_delays_sz
	int *exp_delays_sz = (int*) malloc(freqs_sz*sizeof(int));
	for (size_t i = 0; i < freqs_sz; ++i)
		exp_delays_sz[i] = modal_delays[i].size();

	// exp_delays 2d array
	int dmaxsz = 0;
	for (size_t i = 0; i < freqs_sz; ++i)
		dmaxsz = std::max(dmaxsz, exp_delays_sz[i]);
	float *exp_delays = (float*) malloc(dmaxsz*freqs_sz*sizeof(float));
	for (size_t i = 0; i < modal_delays.size(); ++i)
		for (size_t j = 0; j < modal_delays[i].size(); ++j)
			exp_delays[i*dmaxsz + j] = modal_delays[i][j];

	
	int n_layers = depths_d.size();
	float *depths = (float*) malloc(n_layers*sizeof(float));
	for (int i=0; i<n_layers; ++i)
		depths[i] = depths_d[i];

	int *Ns_points = (int*) malloc(n_layers*sizeof(int));
	for (int i=0; i<n_layers; ++i)
		Ns_points[i] = Ns_points_d[i];

	float *residuals = (float*) malloc(sz*sizeof(float));

	m_CopyToGPU2(R,   sz, float);
	m_CopyToGPU2(tau, sz, float);
	m_CopyToGPU2(rhob,sz, float);
	m_CopyToGPU2(cb,  sz, float);
	m_CopyToGPU2(cws, sz*cws_sz, float);
	m_CopyToGPU2(freqs, freqs_sz, float);
	m_CopyToGPU2(exp_delays, dmaxsz*freqs_sz, float);
	m_CopyToGPU2(exp_delays_sz, freqs_sz, int);
	m_CopyToGPU2(depths, n_layers, float);
	m_CopyToGPU2(Ns_points, n_layers, int);
	m_CopyToGPU2(residuals, sz, float);

	hipEvent_t kernel_start, kernel_stop;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	hipEventRecord(kernel_start);

	EvalPoints_gpukernel <<< sz/(1<<BLOCKSIZE), 1<<BLOCKSIZE >>> 
		(sz, cws_sz, dmaxsz, g_cws, g_Ns_points, g_depths, g_R, g_tau, g_rhob, g_cb, g_freqs, freqs_sz,
			g_exp_delays, g_exp_delays_sz, g_residuals);
	#ifndef NDEBUG
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	checkCudaErrors(err);
	printf("\n Bla");
	#endif

	hipDeviceSynchronize();
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	float runTime;
	hipEventElapsedTime(&runTime, kernel_start, kernel_stop);


	float tm = runTime / 1000;
	printf("\n Time: %f", tm);
	
	checkCudaErrors(hipMemcpy((void*) residuals, (void*)g_residuals, 
				sz*sizeof(float), hipMemcpyDeviceToHost));
	for (size_t i = 0; i < sz; ++i)
		batch[i].residual = residuals[i];

	m_FreeHostAndGPU(Ns_points);
	m_FreeHostAndGPU(depths);
	m_FreeHostAndGPU(exp_delays_sz);
	m_FreeHostAndGPU(exp_delays);
	m_FreeHostAndGPU(freqs);
	m_FreeHostAndGPU(R);
	m_FreeHostAndGPU(tau);
	m_FreeHostAndGPU(rhob);
	m_FreeHostAndGPU(cb);
	m_FreeHostAndGPU(cws);
	m_FreeHostAndGPU(residuals);

}

search_space_point sspemdd_sequential::generateRandomPoint()
{
	std::vector <unsigned> point_indexes;
	for (const auto &var :search_space)
	{
		std::uniform_int_distribution<int> uni(0, var.size()-1);
		point_indexes.push_back(uni(rng));
		//std::cout << " rand var " << randnum << std::endl;
	}
	search_space_point point = fromPointIndexesToPoint(point_indexes);
	return std::move(point);
}

void sspemdd_sequential::ILSGPU(int ils_runs)
{
	std::cout << "Start ILS GPU" << std::endl;

	const size_t batch_size = 4*1280;
	search_space_point global_record;

	std::cout << "Global record" << global_record.residual << std::endl;
	for (size_t i = 0; i < ils_runs; ++i)
	{
		std::vector <search_space_point> batch;
		for (size_t j = 0; j < batch_size; ++j)
			batch.push_back(generateRandomPoint());
		std::cout << " start eval " << std::endl;
		//for (auto &point: batch) EvalPointGPU(point, freqs, Ns_points, depths, modal_delays);
		EvalPointBatchGPU(batch, freqs, Ns_points, depths, modal_delays);
		search_space_point best = *std::min_element(std::begin(batch), std::end(batch));
		std::cout << "Best of batch: " << best.residual << std::endl;
		if (best < global_record )
			global_record = best;
	}
	record_point = global_record;
	std::cout << "Global record" << global_record.residual << std::endl;
}

